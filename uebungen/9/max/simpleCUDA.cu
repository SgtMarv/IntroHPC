#include "hip/hip_runtime.h"
#include <iostream> 
#include <stdlib.h>
#include <sys/time.h>
#include <cstring>
#include <math.h>

using namespace std;

double time_diff(timeval a, timeval b){
    return (b.tv_sec-a.tv_sec) * pow(10,-6) * (b.tv_usec - a.tv_usec);
}

void init_array(float* ary, int n){
    for(int i = 0; i<n; i++){
        ary[i] = (float)rand()/(float(RAND_MAX));
    }
}



void saxpy_cpu(float* x, float* y, float a, int n){
    for(int i=0; i<n; i++){
        y[i] = a * x[i] + y[i];
    }
}


__global__ void saxpy_gpu(float* x, float* y, float a, int n){

    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if ( i<n ) {
        y[i] = a * x[i] + y[i];
    }

}


int main (int argc, char **argv){

    long n;      //problem size
    float a = 2.0; //alpha factor in equation, hardcoded
    bool pinned_mem; //use pinned memory or not

	long factor = 1; //for command line parsing
	char *pos = NULL;

  if (argc != 4) {
    //sprintf ("Usage: %s <problem size{k,M,G}> <block size>\n", argv[0]);
    exit (0);
  }
  pos = strrchr (argv[1], 'k');
  if (pos != NULL) {
  	factor = 1024;
  	*pos = '\0'; //terminate input string here
  }
  pos = strrchr (argv[1], 'M');
  if (pos != NULL) {
  	factor = 1024*1024;
  	*pos = '\0'; //terminate input string here
  }
  pos = strrchr (argv[1], 'G');
  if (pos != NULL) {
  	factor = 1024*1024*1024;
  	*pos = '\0'; //terminate input string here
  }
  n = atol (argv[1]);
  n *= factor;

	pinned_mem = atoi(argv[3]);
     
  long numThreadsPerBlock;
  int selectedDevice = 0;
  
  numThreadsPerBlock = atol (argv[2]);
  int numBlocks = (n+numThreadsPerBlock-1) / numThreadsPerBlock;
     
  if (numThreadsPerBlock > 1024) {
    //printf ("ERROR: numThreadsPerBlock must be <= 1024!\n");
	cout << "Err: numThready <= 1024 " << endl;
    return 0;
  }
  if (numBlocks >= 65536) {
    //printf ("ERROR: numBlocks must be < 65536 (is %ld)!\n", numBlocks);
	cout << "Err: numBlocks < 65536" << endl;
    return 0;
  }

    //for timing
    timeval start, stop;
    double t_init; 
    double t_copy;
    double t_cpu;
    double t_gpu;
    double t_back;


  /////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    //fprintf(stderr, "Sorry, no CUDA device fount");
    return 1;
  }
  if (selectedDevice >= deviceCount) {
    //fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    return 1;
  }
  hipSetDevice(selectedDevice);
  hipDeviceSynchronize();


    //allcoate mem
    float* x;
    float* y;
    float* d_x;
    float* d_y;

    //where to alloc host vars
    if (pinned_mem){
        hipHostMalloc((void**) &x, n*sizeof(float));
        hipHostMalloc((void**) &y, n*sizeof(float));
    }
    else {
        x = (float*) malloc(n*sizeof(float));
        y = (float*) malloc(n*sizeof(float));
    }

    //allocate device vars on GPU 
    hipMalloc((void**)&d_x, n*sizeof(float));
    hipMalloc((void**)&d_y, n*sizeof(float));

   
    //init arrays on CPU
    gettimeofday(&start, NULL);
    init_array(x, n);
    init_array(y, n);
    gettimeofday(&stop, NULL);
    t_init = time_diff(start,stop);

    
    // copy to GPU
    gettimeofday(&start, NULL);
    hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n*sizeof(float), hipMemcpyHostToDevice);
    gettimeofday(&stop, NULL);
    t_copy = time_diff(start, stop);

    // do operation on CPU 
    gettimeofday(&start, NULL);
    saxpy_cpu(x, y, a, n);
    gettimeofday(&stop, NULL);
    t_cpu = time_diff(start, stop);

    // do operation on GPU
    gettimeofday(&start, NULL);
    saxpy_gpu<<<numBlocks, numThreadsPerBlock>>>(d_x, d_y, a, n);
    hipDeviceSynchronize();
    gettimeofday(&stop, NULL);
    t_gpu = time_diff(start, stop);

    
    //write back and compare
    float* tmp_y; 
    if (pinned_mem){
        hipHostMalloc((void**) &tmp_y, n*sizeof(float));
    }
    else {
        tmp_y = (float*) malloc(n*sizeof(float));
    }

    gettimeofday(&start, NULL);
    hipMemcpy(tmp_y, d_y, n*sizeof(float), hipMemcpyDeviceToHost);
    gettimeofday(&stop, NULL);
    t_back = time_diff(start, stop);

    int err_count = 0;
    for(int i = 0; i<n; i++){
        if(abs(tmp_y[i]-y[i]) > 1e-6){
            cout << "Error on comparison on index: " << i << endl;
            err_count++;
        }
    }
    cout << "Error count: " << err_count << endl;


    //cleanup
    if(pinned_mem){
        hipHostFree(x);
        hipHostFree(y);
        hipHostFree(tmp_y);
    }
    else{
        free(x);
        free(y);
        free(tmp_y);
    }

    hipFree(d_x);
    hipFree(d_y);

    //report timing
    cout << "Initialization: " << t_init << " s\n";
    cout << "Copy to GPU: " << t_copy << " s\n";
    cout << "Sequential CPU: " << t_cpu << " s\n";
    cout << "Parallel GPU: " << t_gpu << " s\n";
    cout << "Writeback: " << t_back << " s\n";


    return 0;



}







