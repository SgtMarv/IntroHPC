#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstring>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>


using namespace std;

static int selectedDevice = 0;
long numThreadsPerBlock;

//copy from ex9 file
void checkErrors(char *label)
{
  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    char *e = (char*) hipGetErrorString(err);
    cout << "Cuda Error: " << e << " (at " << label <<")" << endl;
  }

  err = hipGetLastError();
  if (err != hipSuccess) {
    char *e = (char*) hipGetErrorString(err);
    cout << "Cuda Error: " << e << " (at " << label <<")" << endl;
  }
}

void synchronize()
{
  hipError_t err;
     
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    char *e = (char*) hipGetErrorString(err);
    cout << "Cuda Error: " << e << " (at synchronize)" << endl;
  }
}
////////////////////////////////////////

double time_diff(timeval a, timeval b){
    return (b.tv_sec-a.tv_sec)+pow(10,-6)*(b.tv_usec-a.tv_usec);
}


void init_mat(float* mat, int size, int init){
    if(init==0){
        for(int i = 0; i<(size*size); i++){
            mat[i] = 0.0;
        }
    } 
    if(init == 1){
        for(int i = 0; i<size; i++){
            for(int j = 0; j<size; j++){
                mat[i*size+j] = (i+1)+(j+1);
            }
        }
    }
    if(init == 2){
        for(int i = 0; i<size; i++){
            for(int j = 0; j<size; j++){
                mat[i*size+j] = (i+1)*(j+1);
            }
        }
    }
}

void print_mat(float* mat, int size){
    if(size>10){
        cout << "Matrix too large, will not print." << endl;
    }
    else{
        for(int i = 0; i<(size*size); i++){
            cout << mat[i] << "\t";
            if((i+1)%size == 0){
                cout << endl;
            }
        }
        cout << endl;
    }
}

void mat_mult_cpu(float* a, float* b, float* c, int size){
    for (int i = 0; i<size; i++){
        for (int k = 0; k<size; k++){
            for (int j = 0; j<size; j++){
                c[i*size+j] += a[i*size+k] * b[k*size+j];
            }
        }
    }
}

__global__ void mat_mult_gpu(float* a, float* b, float* c, int size){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float pval = 0.0;
    if(row<size&&col<size){
for (int k = 0; k< size; k++){
        pval += a[row * size + k] * b[k * size + col];
    }

    c[row*size+col] = pval;
}
}

__global__ void mat_mult_shared(float* a, float* b, float* c, int size){

	__shared__ float mem_ds [];

	 float *a_s=&(mem_ds [0]);
	float *b_s=&(mem_ds [size*size]);
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float pval = 0.0;
    if(row<size&&col<size){
	  for ( int m = 0; m < size; ++m ) {
     // Collaborative loading of a and b tiles into shared memory
     a_s [row*size+col] = a [ row * size + ( m * size + col ) ];
	 b_s [row*size + col] = b [ col + ( m * size	 + row ) * size ];
 __syncthreads();
 for ( int k = 0; k < size; ++k )
 	pval += a_s[row*size+k] * b_s[k*size+col];
 __syncthreads ();
 }
     c[row*size+col] = pval;
}
}
    

int main(int argc, char** argv){

    int size;
    float t_send;
    size = atoi(argv[1]);

    numThreadsPerBlock = atol(argv[2]);
    int numBlocks = 1;//(size+numThreadsPerBlock-1)/numThreadsPerBlock;
    if (numThreadsPerBlock > 1024){
        cout << "ERROR: NumThreadPerBlock must be < 1024" << endl;
        return 0;
    }
    if(numBlocks >65536){
        cout << "ERROR: numBlocks must be < 65536, is " << numBlocks << endl;
        return 0;
    }

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount== 0){
        cout << "ERROR: No device found" << endl;
        return 0;
    }
    if(selectedDevice >= deviceCount){
        cout << "ERROR: Choose dev ID between 0 and " << deviceCount-1 << endl;
        return 0;
    }
    hipSetDevice(selectedDevice);
    checkErrors("init");


    int seed = time(NULL);
    srand(seed);

    timeval start,stop;

    float* h_a = new float [size*size];
    float* h_b = new float [size*size];
    float* h_c = new float [size*size];
    float* c_comp = new float [size*size];

    float* d_a = new float [size*size];
    float* d_b = new float [size*size];
    float* d_c = new float [size*size];

    init_mat(h_a,size,1);
    init_mat(h_b,size,2);
    init_mat(h_c,size,0);

    hipMalloc((void**) &d_a, size*size*sizeof(float));
    hipMalloc((void**) &d_b, size*size*sizeof(float));
    hipMalloc((void**) &d_c, size*size*sizeof(float));
    checkErrors("mem alloc");

    gettimeofday(&start,NULL);	
    hipMemcpy(d_a, h_a, size*size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size*size*sizeof(float), hipMemcpyHostToDevice);
    gettimeofday(&stop,NULL);
    t_send=time_diff(start,stop);
    checkErrors("copy date to dev");
   

    print_mat(h_a,size);
    print_mat(h_b,size);

    //mat mult CPU
    gettimeofday(&start,NULL);
    mat_mult_cpu(h_a,h_b,h_c,size);
    gettimeofday(&stop,NULL);
    cout << "Time for CPU: " << time_diff(start,stop) << " s" << endl;

    dim3 dimBlock(size,size);
    //mat mult GPU
    gettimeofday(&start,NULL);
    mat_mult_gpu<<<numBlocks,dimBlock>>> (d_a,d_b,d_c,size);
    gettimeofday(&stop,NULL);
    synchronize();
    cout << "Time for GPU: " << time_diff(start,stop) << " s" << endl;
    checkErrors("compute on GPU");

   //mat mult with shared mem on gpu
	int size_block=size;
 	int shared_size=2*size_block*size_block*sizeof(float);
	gettimeofday(&start,NULL);

	mat_mult_shared<<numBlocks,dimBlock,shared_size>> (d_a,d_b,d_c,size);
	gettimeofday(&stop,NULL);
  	synchronize();
	cout <<"Time with shared memory: "<< time_diff(start,stop)<<endl;	
	checkErrors("compute on GPU_shared");

    gettimeofday(&start,NULL);
    hipMemcpy(c_comp, d_c, size*size*sizeof(float), hipMemcpyDeviceToHost); 
    gettimeofday(&stop,NULL);    
    t_send+=time_diff(start,stop);
    checkErrors("copy back");
    cout <<"Time for Sending: "<<t_send<<endl;

    print_mat(h_c,size);
	print_mat(c_comp,size);
	for(int i=0;i<size;i++){
	for(int j=0;j<size;j++){
		if(h_c[i*size+j]!=c_comp[i*size+j])cout<<"e";}}
	
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
   
    free(h_a);
    free(h_b);
    free(h_c);
    free(c_comp);

    return 0;
}
