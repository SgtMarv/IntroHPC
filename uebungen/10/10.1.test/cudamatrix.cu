#include "hip/hip_runtime.h"
/*
 *
 * nullKernelAsync.cu
 *
 * Microbenchmark for throughput of asynchronous kernel launch.
 *
 * Build with: nvcc -I ../chLib <options> nullKernelAsync.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
*/

#include <stdio.h>

#include "chTimer.h"

__global__ void Matrixmulti(float A[N][N], float B[N][N], float C[N][N])
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y; 
	
	
	for (int k=0;k<N;k++)
	{
	 C[i][j]+=A[i][k]*B[k][j];
	}}

}

int
main( int argc, char *argv[] )
{ 
	int matrixsize N=16;
	
	void *dmemA = hipMalloc ( N*N*sizeof ( float ) ); // Allocate GPU memory
	void *dmemB = hipMalloc ( N*N*sizeof ( float ) ); // Allocate GPU memory
	void *dmemC = hipMalloc ( N*N*sizeof ( float ) ); // Allocate GPU memory
	
    void *hmemA = malloc ( N*N*sizeof ( float ) ); // Allocate CPU memory
	void *hmemB = malloc ( N*N*sizeof ( float ) ); // Allocate CPU memory
	void *hmemC = malloc ( N*N*sizeof ( float ) ); // Allocate CPU memory
	
	
	
	
	hipMemcpy ( dmemA, hmemA, N*N*sizeof ( float ), hipMemcpyHostToDevice ); 
	hipMemcpy ( dmemB, hmemB, N*N*sizeof ( float ), hipMemcpyHostToDevice ); 
	
    printf( "Measuring... " ); fflush( stdout );

    chTimerTimestamp start, stop;

	dim3 dimBock(N,N);
	
	
    chTimerGetTime( &start );
    
        matrixmulti<<<1,dimBlock>>>(dmemA,dmemB,dmemC,N);
    
    hipDeviceSynchronize();
    chTimerGetTime( &stop );

    {
        double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
        double usPerLaunch = microseconds / (float) cIterations;

        printf( "%.2f us\n", usPerLaunch );
    }
	
	hipMemcpy ( hmemC, dmemC, N*N*sizeof ( float ), hipMemcpyDeviceToHost ); 
	
	
	cudafree(dmemA);
	cudafree(dmemB);
	cudafree(dmemC);
	free(hmemA);
	free(hmemB);
	free(hmemC);
    return 0;
}
