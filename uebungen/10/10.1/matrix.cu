#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstring>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>


using namespace std;

static int selectedDevice = 0;
long numThreadsPerBlock;

//copy from ex9 file
void checkErrors(char *label)
{
  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    char *e = (char*) hipGetErrorString(err);
    cout << "Cuda Error: " << e << " (at " << label <<")" << endl;
  }

  err = hipGetLastError();
  if (err != hipSuccess) {
    char *e = (char*) hipGetErrorString(err);
    cout << "Cuda Error: " << e << " (at " << label <<")" << endl;
  }
}

void synchronize()
{
  hipError_t err;
     
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    char *e = (char*) hipGetErrorString(err);
    cout << "Cuda Error: " << e << " (at synchronize)" << endl;
  }
}
////////////////////////////////////////

double time_diff(timeval a, timeval b){
    return (b.tv_sec-a.tv_sec)+pow(10,-6)*(b.tv_usec-a.tv_usec);
}


void init_mat(float* mat, int size, int init){
    if(init==0){
        for(int i = 0; i<(size*size); i++){
            mat[i] = 0.0;
        }
    } 
    if(init == 1){
        for(int i = 0; i<size; i++){
            for(int j = 0; j<size; j++){
                mat[i*size+j] = (i+1)+(j+1);
            }
        }
    }
    if(init == 2){
        for(int i = 0; i<size; i++){
            for(int j = 0; j<size; j++){
                mat[i*size+j] = (i+1)*(j+1);
            }
        }
    }
}

void print_mat(float* mat, int size){
    if(size>10){
        cout << "Matrix too large, will not print." << endl;
    }
    else{
        for(int i = 0; i<(size*size); i++){
            cout << mat[i] << "\t";
            if((i+1)%size == 0){
                cout << endl;
            }
        }
        cout << endl;
    }
}

void mat_mult_cpu(float* a, float* b, float* c, int size){
    for (int i = 0; i<size; i++){
        for (int k = 0; k<size; k++){
            for (int j = 0; j<size; j++){
                c[i*size+j] += a[i*size+k] * b[k*size+j];
            }
        }
    }
}

__global__ void mat_mult_gpu(float* a, float* b, float* c, int size){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float pval = 0.0;
    for (int k = 0; k< size; k++){
        pval += a[row * size + k] * b[k * size + col];
    }

    c[row*size+col] = pval;

}
    

int main(int argc, char** argv){

    int size;
    size = atoi(argv[1]);

    numThreadsPerBlock = atol(argv[2]);
    int numBlocks = (size+numThreadsPerBlock-1)/numThreadsPerBlock;
    if (numThreadsPerBlock > 1024){
        cout << "ERROR: NumThreadPerBlock must be < 1024" << endl;
        return 0;
    }
    if(numBlocks >65536){
        cout << "ERROR: numBlocks must be < 65536, is " << numBlocks << endl;
        return 0;
    }

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount== 0){
        cout << "ERROR: No device found" << endl;
        return 0;
    }
    if(selectedDevice >= deviceCount){
        cout << "ERROR: Choose dev ID between 0 and " << deviceCount-1 << endl;
        return 0;
    }
    hipSetDevice(selectedDevice);
    checkErrors("init");


    int seed = time(NULL);
    srand(seed);

    timeval start,stop;

    float* a = new float [size*size];
    float* b = new float [size*size];
    float* c = new float [size*size];
    float* c_comp = new float [size*size];

    float* d_a = new float [size*size];
    float* d_b = new float [size*size];
    float* d_c = new float [size*size];

    init_mat(a,size,1);
    init_mat(b,size,2);
    init_mat(c,size,0);

    hipMalloc((void**) &d_a, size*size*sizeof(float));
    hipMalloc((void**) &d_b, size*size*sizeof(float));
    hipMalloc((void**) &d_c, size*size*sizeof(float));
    checkErrors("mem alloc");

    hipMemcpy(d_a, a, size*size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size*size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size*size*sizeof(float), hipMemcpyHostToDevice);
    checkErrors("copy date to dev");


    print_mat(a,size);
    print_mat(b,size);

    //mat mult CPU
    gettimeofday(&start,NULL);
    mat_mult_cpu(a,b,c,size);
    gettimeofday(&stop,NULL);
    cout << "Time for CPU: " << time_diff(start,stop) << " s" << endl;

    //mat mult GPU
    gettimeofday(&start,NULL);
    mat_mult_gpu<<<numBlocks, numThreadsPerBlock>>> (d_a,d_b,d_c,size);
    gettimeofday(&stop,NULL);
    synchronize();
    cout << "Time for GPU: " << time_diff(start,stop) << " s" << endl;
    checkErrors("compute on GPU");

    hipMemcpy(c_comp, d_c, size*size*sizeof(float), hipMemcpyDeviceToHost); 
    checkErrors("copy back");

    print_mat(c,size);


    return 0;
}
