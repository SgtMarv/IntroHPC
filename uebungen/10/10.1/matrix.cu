#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <math.h>


using namespace std;

static int selectedDevice = 0;
long numThreadsPerBlock;

//copy from ex9 file
void checkErrors(char *label)
{
  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    char *e = (char*) hipGetErrorString(err);
    cout << "Cuda Error: " << e << " (at " << label <<")" << endl;
  }

  err = hipGetLastError();
  if (err != hipSuccess) {
    char *e = (char*) hipGetErrorString(err);
    cout << "Cuda Error: " << e << " (at " << label <<")" << endl;
  }
}

void synchronize()
{
  hipError_t err;
     
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    char *e = (char*) hipGetErrorString(err);
    cout << "Cuda Error: " << e << " (at synchronize)" << endl;
  }
}
////////////////////////////////////////


void init_mat(float* mat, int size, int init){
    if(init==0){
        for(int i = 0; i<(size*size); i++){
            mat[i] = 0.0;
        }
    } 
    if(init == 1){
        for(int i = 0; i<size; i++){
            for(int j = 0; j<size; j++){
                mat[i*size+j] = (i+1)+(j+1);
            }
        }
    }
    if(init == 2){
        for(int i = 0; i<size; i++){
            for(int j = 0; j<size; j++){
                mat[i*size+j] = (i+1)*(j+1);
            }
        }
    }
}

void print_mat(float* mat, int size){
    if(size>10){
        cout << "Matrix too large, will not print." << endl;
    }
    else{
        for(int i = 0; i<(size*size); i++){
            cout << mat[i] << "\t";
            if((i+1)%size == 0){
                cout << endl;
            }
        }
        cout << endl;
    }
}

void mat_mult_cpu(float* a, float* b, float* c, int size){
    for (int i = 0; i<size; i++){
        for (int k = 0; k<size; k++){
            for (int j = 0; j<size; j++){
                c[i*size+j] += a[i*size+k] * b[k*size+j];
            }
        }
    }
}

__global__ void mat_mult_gpu(float* a, float* b, float* c, int size){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float pval = 0.0;
    for (int k = 0; k< size; k++){
        pval += a[row * size + k] * b[k * size + col];
    }

    c[row*size+col] = pval;

}
    

int main(int argc, char** argv){

    int size;
    size = atoi(argv[1]);

    numThreadsPerBlock = atol(argv[2]);
    int numBlocks = (size+numThreadsPerBlock-1)/numThreadsPerBlock;
    if (numThreadsPerBlock > 1024){
        cout << "ERROR: NumThreadPerBlock must be < 1024" << endl;
        return 0;
    }
    if(numBlocks >65536){
        cout << "ERROR: numBlocks must be < 65536, is " << numBlocks << endl;
        return 0;
    }

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount== 0){
        cout << "ERROR: No device found" << endl;
        return 0;
    }
    if(selectedDevice >= deviceCount){
        cout << "ERROR: Choose dev ID between 0 and " << deviceCount-1 << endl;
        return 0;
    }
    hipSetDevice(selectedDevice);
    checkErrors("init");


    int seed = time(NULL);
    srand(seed);

    timeval start,stop;

    float* a = new float [size*size];
    float* b = new float [size*size];
    float* c = new float [size*size];

    init_mat(a,size,1);
    init_mat(b,size,2);
    init_mat(c,size,0);

    print_mat(a,size);
    print_mat(b,size);

    mat_mult_cpu(a,b,c,size);

    print_mat(c,size);


    return 0;
}
